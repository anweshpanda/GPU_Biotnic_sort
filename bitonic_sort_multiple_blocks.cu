#include "hip/hip_runtime.h"
%%cu
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

int no_blocks=64;
int no_threads=1024;
int total_size=no_blocks*no_threads;

void sequential_sort(float* a ,int n)
{
   for(int i=2;i<=n;i=i*2)
    {
      for(int j=i/2;j>0;j=j/2)
      {
        for(int index =0 ;index<n;index++)
         { 
          int k=index^j;
            if(index<k)
            {
                if((i&index) )
                {
                    if(a[k]>a[index])
                    {
                        float tmp=a[k];
                         a[k]=a[index];
                         a[index]=tmp;
                    }
                }
                else
                {
                    if(a[k]<a[index])
                    {
                        float tmp=a[k];
                        a[k]=a[index];
                        a[index]=tmp;
                    }
                }
                     
             }
           }
        }
    }
}


__global__ void sort(float *glob_arr,int n) 
{
    int index=threadIdx.x ;
    int glob_index=threadIdx.x + blockDim.x * blockIdx.x;
 


    extern __shared__ int a[1024];
    a[index]=glob_arr[glob_index];
    
    for(int i=2;i<=1024;i=i*2)
    {
      for(int j=i/2;j>0;j=j/2)
      {
          int k=index^j;
          if(index<k)
          {
            if((i&index) )
              {
                if(a[k]>a[index])
                {
                  float tmp=a[k];
                  a[k]=a[index];
                  a[index]=tmp;
                }
              }
          else
              {
                if(a[k]<a[index])
                {
                  float tmp=a[k];
                  a[k]=a[index];
                  a[index]=tmp;
                }
              }
            }
        __syncthreads();
    }
  } 
 glob_arr[glob_index]=a[index]; 
 
 }

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
 __global__ void merge(float *glob_arr,float* final_arr,int n) 
{
   
    int glob_index=threadIdx.x + blockDim.x * blockIdx.x;
 
 int i = glob_index;
 float key =glob_arr[i];
 int pos =0;
 int offset =1024;  
 int x = (i/offset)*offset;

 for (int j = 0; j < n; j+=offset)
 {
  
  int s=j;
  int e=j+offset-1;
  
  
   if(j<x)
   {
    while(s<=e)
    {
      int mid =(s+e)/2;
      if(glob_arr[mid] == key)
      {
       
        s=mid+1;
      }
      else if(glob_arr[mid]>key)
      {
        e = mid -1;
      }
      else
      {
        s = mid+1;
      }
    }
    pos += (s-j);
   }
   if(j>x)
   {
    while(s<=e)
    {
      int mid =(s+e)/2;
      if(glob_arr[mid] ==key)
      {
       
        e = mid -1;
      }
      else if(glob_arr[mid]>key)
      {
        e=mid-1;
      }
      else
      {
        s = mid+1;
      }
    }
    pos += (s-j);

   }
}
 pos += (i-x);
 final_arr[pos] = key;
 }


 //////////////////////////////////////////////////////////////////////////////////////////
int main() 
{
  double average_time1 = 0;
  double average_time2 = 0;
  double average_time3 = 0;

  for (int j = 0; j <10 ; ++j)
  {
       float* a = (float*)malloc(total_size*sizeof(float));
       float* arr = (float*)malloc(total_size*sizeof(float));
       struct timespec start1,start2,start3,end1,end2,end3;
       

       for(int i=0;i<total_size;i++)
       {
           a[i]=rand();
           arr[i] =a[i];
       }
       clock_gettime( CLOCK_REALTIME,&start1);
       sequential_sort(arr,total_size);
       clock_gettime( CLOCK_REALTIME,&end1);

       for (int i = 1; i < total_size; ++i)
       {
         if(arr[i]<arr[i-1])
         {
          printf("wrong answer\n");
         }
       }


       int size = sizeof(float);
       float *d_a,*d_b;
       hipMalloc((void **)&d_a, size*total_size);
       hipMalloc((void **)&d_b, size*total_size);

       clock_gettime( CLOCK_REALTIME,&start2);

       hipMemcpy(d_a, a, size*total_size, hipMemcpyHostToDevice);

       clock_gettime( CLOCK_REALTIME,&start3);

       sort<<<no_blocks,no_threads>>>(d_a,total_size);
       merge<<<no_blocks,no_threads>>>(d_a,d_b,total_size);
       clock_gettime( CLOCK_REALTIME,&end3);

      hipMemcpy(a, d_b, size*total_size, hipMemcpyDeviceToHost);

       clock_gettime( CLOCK_REALTIME,&end2);
       

      for(int i=0;i<total_size;i++)
      {
        if(arr[i]!=a[i])
        {
            printf("wrong answer\n");
        }
      }
       
      double elapsed1 = (end1.tv_sec-start1.tv_sec)*1000000000 + end1.tv_nsec-start1.tv_nsec;
      double elapsed2 = (end2.tv_sec-start2.tv_sec)*1000000000 + end2.tv_nsec-start2.tv_nsec;
      double elapsed3 = (end3.tv_sec-start3.tv_sec)*1000000000 + end3.tv_nsec-start3.tv_nsec;
      average_time1 += elapsed1;
      average_time3 += elapsed3;
      average_time2 += elapsed2;


       
       


       hipFree(d_a);
       hipFree(d_b);
  }
    printf("time taken for sequential sorting\n");
    printf("%lf\n",average_time1/10);

    printf("time taken for gpu sorting without memcopy\n");
    printf("%lf\n",average_time3/10);

    printf("time taken for gpu sorting with memcopy\n");
    printf("%lf\n",average_time2/10);

    printf("Speed up achieved is\n");
    printf("%lf\n",(average_time1/average_time2));

    printf("Speed up achieved without memcopy is\n");
    printf("%lf\n",(average_time1/average_time3));



 return 0;
}
